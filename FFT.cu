
#include <hip/hip_runtime.h>
//#include "Field.hpp"

//#include "cuda_runtime.h"
//#include "device_launch_parameters.h"

//we are computing FFT of the size 2**N 
#define N 1024
#define LOG_N 10

/*using namespace arithmetic;

std::initializer_list<uint_default_t> p = { 0x2370fb049d410fbe, 0x4e761a9886e50241, 0x7d023f4018000001, 0x7e80600000000001 };
static const bignum<MAX_BITSIZE> modulus(p);

using field = Field<MAX_BITSIZE, modulus>;

__constant__ field dev_roots_of_unity[LOG_N];

__global__ void FFTKernel(field* input_buf, field* output_buf);

int main()
{
	//first find suitable Cuda device
	//TBD: or split between several CUDA devices if possible
	int device_count;
	cudaError_t cudaStatus = cudaGetDeviceCount(&device_count);
	if (cudaStatus != cudaSuccess) {
		fprintf(stderr, "cudaGetDeviceCount failed!");
		return 1;
	}
	if (device_count == 0)
	{
		fprintf(stderr, "No suitable CUDA devices were found!");
		return 1;
	}

	cudaDeviceProp prop;
	cudaStatus = cudaGetDeviceProperties(&prop, 0);

	if (cudaStatus != cudaSuccess) {
		fprintf(stderr, "cudaGetDeviceCount failed!");
		return 1;
	}

	//TODO: check if there are enough constant memory and other additional checks
	//set appropriate device
	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = cudaSetDevice(0);
	if (cudaStatus != cudaSuccess)
	{
		fprintf(stderr, "cudaSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	field input[N];
	field output[N];

	//we are goint to calculate roots of unity in corresponding powers
	field roots_of_unity[LOG_N];
	//TODO: compute them in advance before the kernel starts

	for (size_t i = 0; i < N; i++)
		input[i] = field::random();

	field* dev_input = nullptr;
	field* dev_output = nullptr;

	// Allocate GPU buffers for three vectors (one input, one output)    .
	cudaStatus = cudaMalloc((void**)&dev_input, N * sizeof(field));
	if (cudaStatus != cudaSuccess)
	{
		fprintf(stderr, "cudaMalloc failed!");
		goto Error;
	}

	cudaStatus = cudaMalloc((void**)&dev_output, N * sizeof(field));
	if (cudaStatus != cudaSuccess)
	{
		fprintf(stderr, "cudaMalloc failed!");
		goto Error;
	}

	// Copy input vector from host memory to GPU buffers.
	cudaStatus = cudaMemcpy(dev_input, input, N * sizeof(field), cudaMemcpyHostToDevice);
	if (cudaStatus != cudaSuccess)
	{
		fprintf(stderr, "cudaMemcpy failed!");
		goto Error;
	}

	//copy precomputed roots of unity to constant memory
	cudaMemcpyToSymbol(dev_roots_of_unity, roots_of_unity, LOG_N * sizeof(field));

	// Launch a kernel on the GPU with one thread for each element.
	FFTKernel << <1, size >> > (dev_input, dev_output);

	// Check for any errors launching the kernel
	cudaStatus = cudaGetLastError();
	if (cudaStatus != cudaSuccess)
	{
		fprintf(stderr, "FFTKernel launch failed: %s\n", cudaGetErrorString(cudaStatus));
		goto Error;
	}

	// cudaDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = cudaDeviceSynchronize();
	if (cudaStatus != cudaSuccess)
	{
		fprintf(stderr, "cudaDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = cudaMemcpy(output, dev_output, N * sizeof(field), cudaMemcpyDeviceToHost);
	if (cudaStatus != cudaSuccess)
	{
		fprintf(stderr, "cudaMemcpy failed!");
		goto Error;
	}

Error:
	cudaFree(dev_input);
	cudaFree(dev_output);

	return 0;
}

//NB: we have precomputed the powers of roots of unity and put them into constant memory
__global__ void FFTKernel(const field* input_buf, field* output_buf)
{
	field temp_buf[N];

	const field* in_buf = input_buf;
	field* out_buf = (LOG_N % 2 ? output_buf : temp_buf);

	int thread_idx = threadIdx.x + blockIdx.x * blockDim.x;

	for (size_t i = 0; i < LOG_N; i++)
	{
		int idx = thread_idx;
		while (idx < N)
		{
			idx += blockDim.x * gridDim.x;
		}
	}
}*/



