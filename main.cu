#include "Field.hpp"

using namespace arithmetic;

#define TEST_LIMBSIZE 4

int main(int argc, char* argv[])
{
	std::initializer_list<uint_default_t> a_list =
	{ 0x2370fb049d410fbe, 0x4e761a9886e50241, 0x7d023f4018000001, 0x7e80600000000001 };

	std::string a_str = "2370fb049d410fbe4e761a9886e502417d023f40180000017e80600000000001";

	bignum<TEST_LIMBSIZE> a_from_list(a_list);
	bignum<TEST_LIMBSIZE> a_from_str(a_str);
	bool flag = (a_from_list == a_from_str);

	bignum<TEST_LIMBSIZE> a = a_from_list;

	std::initializer_list<uint_default_t> b_list =
	{ 0xc1d2a1e2bccf0c52, 0x799a533a0366b901, 0xa77df6a97da46ddc, 0x952d97f7a515ce82 };

	bignum<TEST_LIMBSIZE> b(b_list);

	//c = a + b
	bignum<TEST_LIMBSIZE> c("e5439ce75a101c10c8106dd28a4bbb43248035e995a46dde13adf7f7a515ce83");
	flag = (a + b == c);
	
	//d = a - b;
	bignum<TEST_LIMBSIZE> d("619e5921e072036bd4dbc75e837e493fd58448969a5b9224e952c8085aea317f");
	flag = (a - b == d);

	//e = a * b;
	bignum<TEST_LIMBSIZE * 2> 
		e("1ad55655d6162f9d349b4e6f1a711ce49b2680f9bdf2c311b56ad18464ea3de96838b8fa17ed2c2220c4abc1a7e40f115758f8064aca60d1ff9e57f7a515ce82");
	bignum<TEST_LIMBSIZE * 2> product(a * b);
	flag = (e == product);

	//divide by single word 
	bignum<1> f(0x9884d881a5d15350);
	auto first_res = (a / f);
	bignum<TEST_LIMBSIZE> first_div("3b7cd16dc948742f39c4cf1c6f82909d1f1042c30195f1de");
	bignum<1> first_rem(0x7b72285f577b70a1);
	flag = (first_res.first == first_div);
	flag = (first_res.second == first_rem);

	//divide by long word;
	std::initializer_list<uint_default_t> g_list =
		{ 0xbaf556405498149b, 0xf44aa2af656398d0 };
	std::initializer_list<uint_default_t> second_div_list = { 0x30878833aad93fed, 0x5be45072345c562d };
	std::initializer_list<uint_default_t> second_rem_list = { 0x8edc5decab396a48, 0x13bf23a68d684371 };
	bignum<2> g(g_list);
	auto second_res = (a / g);
	bignum<3> second_div(second_div_list);
	bignum<2> second_rem(second_rem_list);
	
		
	flag = (second_res.first == second_div);
	flag = (second_res.second == second_rem);



	return flag;
}